#include <iostream>
#include <hip/hip_runtime.h>

int main(int argc, char** argv) {
	/* manually set device */
	int device_id = 0;
	if (argc > 1) {
		device_id = atoi(argv[1]);
		
	}
	hipSetDevice(device_id);
	
	/* start coding from here */


	hipDeviceReset();
	return 0;
}
